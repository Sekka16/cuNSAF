#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

// 单个block计算出总和
__global__ void SimpleSumReductionKernel(float *input, float *output) {

  int idx = 2 * threadIdx.x;
  
  for (int stride = 1; stride < blockDim.x; stride <<= 1) {
    if (idx % stride == 0)
      input[idx] += input[idx + stride];  
    __syncthreads();
  }

  if (idx == 0) {
    *output = input[0];
  }
}

int main() {
  // 1. 设置blockSize和gridSize
  const int elemNums = 256;
  const int blockSize = 256;
  const int gridSize = (elemNums + blockSize - 1) / blockSize;

  // 2. 分配host端内存
  float *h_input = (float *)malloc(sizeof(float) * elemNums);
  float *h_output = (float *)malloc(sizeof(float) * elemNums);

  srand(time(NULL));
  for (int i = 0; i < elemNums; i++) {
    h_input[i] = (float)rand() / RAND_MAX;
  }

  // 3. 分配device端内存
  float* d_input, *d_output;
  hipMalloc((void **)&d_input, elemNums * sizeof(float));
  hipMalloc((void **)&d_output, sizeof(float));

  // 4. 拷贝host端的input到device端
  hipMemcpy(d_input, h_input, elemNums * sizeof(float), hipMemcpyHostToDevice);

  // 5. 启动内核
  SimpleSumReductionKernel<<<gridSize, blockSize>>>(d_input, d_output);

  // 6. 拷贝device端output到host端
  hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

  std::cout << *h_output << std::endl;

  // 7. 释放内存
  free(h_input);
  free(h_output);
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
