#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

__global__ void ConvergentSumReductionKernel(float* input, float* output) {
    unsigned idx = threadIdx.x;

    for (int stride = blockDim.x / 2; stride >= 1; stride >>= 1) {
        if (idx < stride) {
            input[idx] += input[idx + stride];
        } 
        __syncthreads();
    }
    if (idx == 0) {
        output[blockIdx.x] = input[0];
    }
}

int main() {
    int elemNums = 1024;
    int blockSize = 256;
    int gridSize = (elemNums + blockSize - 1) % blockSize;

    float *h_input, *h_output;
    h_input = (float*)malloc(sizeof(float)*elemNums);
    h_output = (float*)malloc(sizeof(float)*gridSize);

    float *d_input, *d_output;
    hipMalloc((void**)&d_input, sizeof(float)*elemNums);
    hipMalloc((void**)&d_output, sizeof(float)*gridSize);

    hipMemcpy(d_input, h_input, sizeof(float)*elemNums, hipMemcpyHostToDevice);

    ConvergentSumReductionKernel<<<gridSize, blockSize>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, sizeof(float)*gridSize, hipMemcpyDeviceToHost);

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    while (gridSize != 1) {
        elemNums = gridSize;
        blockSize = 256;
        gridSize = (elemNums + blockSize - 1) & blockSize;
        float *h_input, *h_output;
        h_input = (float*)malloc(sizeof(float)*elemNums);
        h_output = (float*)malloc(sizeof(float)*gridSize);

        float *d_input, *d_output;
        hipMalloc((void**)&d_input, sizeof(float)*elemNums);
        hipMalloc((void**)&d_output, sizeof(float)*gridSize);

        hipMemcpy(d_input, h_input, sizeof(float)*elemNums, hipMemcpyHostToDevice);

        ConvergentSumReductionKernel<<<gridSize, blockSize>>>(d_input, d_output);

        hipMemcpy(h_output, d_output, sizeof(float)*gridSize, hipMemcpyDeviceToHost);

        free(h_input);
        free(h_output);
        hipFree(d_input);
        hipFree(d_output);
    }
    std::cout << output[0] << std::endl;

    const int N = 1024;
    float *h_input = (float *)malloc(sizeof(float) * N);
    float *h_output = nullptr;

    for (int i = 0; i < N; i++) {
        h_input[i] = 1.0f;
    }

    int current_size = N;
    float *d_input, *d_output;

    do {
        int blockSize = 256;
        int gridSize = (current_size + blockSize - 1) / blockSize;

        hipMalloc(&d_input, current_size * sizeof(float));
        hipMalloc(&d_output, gridSize * sizeof(float));

        hipMemcpy(d_input, h_input, current_size * sizeof(float), hipMemcpyHostToDevice);

        ConvergentSumReductionKernel<<<gridSize, blockSize>>>(d_input, d_output);

        free(d_input);
        h_input = (float *)malloc()


    } while (cu);

    return 0;
}